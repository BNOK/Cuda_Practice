#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_vector_types.h>

#include <opencv2/opencv.hpp>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>

#include <iostream>
#include <stdio.h>
using namespace cv;
using namespace std;

//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

uchar4* converting_UCHAR_UCHAR4(uchar* input, int size);

void DisplayUchar4(uchar4* arr, int size);
void DisplayUchar(uchar* arr, int size);
void ucharToMat(uchar* p2, int rows, int cols);

__global__ void addKernel(uchar4* a, uchar* b);

int main()
{
    //reading image and getting data 
    string file_path = "C:/Users/Mega-Pc/Desktop/git-project/Cuda_Practice/Another_Desperate_Code/Lena.png";
    Mat img = imread(file_path, -1);
    Mat imageRGBA;
    cvtColor(img, imageRGBA, COLOR_BGR2BGRA);
    

    uchar* imgData = imageRGBA.data;

    imshow("color image", imageRGBA);

    waitKey(0);

    // image dimensions
    int rows = imageRGBA.rows;
    int cols = imageRGBA.cols;
    int channels = imageRGBA.channels();

    printf(" rows : %d , cols : %d , channels : %d \n", rows, cols, channels);

    
    //sizes and numbers or bytes for uchar and uchar4
    int imageSize = rows * cols * channels;
    int BYTE_SIZE = imageSize * sizeof(uchar);
    
    cout << " after : " << endl;
    int imageSize_4 = rows * cols;
    int BYTE_SIZE_4 = imageSize_4 * sizeof(uchar4);
    
    //host output variable
    uchar* out_imgData = (uchar*)malloc(imageSize_4 * sizeof(uchar));

    //input data after converting to uchar4
    uchar4* out = (uchar4*)malloc(BYTE_SIZE_4);
    out = converting_UCHAR_UCHAR4(imgData, imageSize_4);
    //DisplayUchar4(out, imageSize_4);



    //device variables
    uchar4* di_imgData;
    uchar* do_imgData;

    //allocating memory
    hipMalloc(&di_imgData, BYTE_SIZE_4);
    hipMalloc(&do_imgData, BYTE_SIZE / channels);
   
    //copying content
    hipMemcpy(di_imgData, out, BYTE_SIZE_4, hipMemcpyHostToDevice);

    //configuring kernel
    dim3 GridSize(16,16, 1);
    dim3 BlockSize(rows /16, cols/16, 1);
    addKernel << <GridSize, BlockSize >> > (di_imgData, do_imgData);

    //returning the results
    hipMemcpy(out_imgData, do_imgData, BYTE_SIZE / channels, hipMemcpyDeviceToHost);

    //printing results
    //namedWindow("color image", WINDOW_AUTOSIZE);
   /* for (int i = 0; i < imageSize_4; i++) {
        printf("out[%d] = %d \n", i, out_imgData[i]);
    }*/
       

    // Show the image inside it.
    //DisplayUchar(out_imgData, imageSize_4);
    ucharToMat(out_imgData,rows,cols);
    



    // Wait for a keystroke.   
    waitKey(0);

    // Destroys all the windows created                         
    destroyAllWindows();
    

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
//{
//    int *dev_a = 0;
//    int *dev_b = 0;
//    int *dev_c = 0;
//    hipError_t cudaStatus;
//
//    // Choose which GPU to run on, change this on a multi-GPU system.
//    cudaStatus = hipSetDevice(0);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
//        goto Error;
//    }
//
//    // Allocate GPU buffers for three vectors (two input, one output)    .
//    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    // Copy input vectors from host memory to GPU buffers.
//    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//    // Launch a kernel on the GPU with one thread for each element.
//    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);
//
//    // Check for any errors launching the kernel
//    cudaStatus = hipGetLastError();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
//        goto Error;
//    }
//    
//    // hipDeviceSynchronize waits for the kernel to finish, and returns
//    // any errors encountered during the launch.
//    cudaStatus = hipDeviceSynchronize();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//        goto Error;
//    }
//
//    // Copy output vector from GPU buffer to host memory.
//    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//Error:
//    hipFree(dev_c);
//    hipFree(dev_a);
//    hipFree(dev_b);
//    
//    return cudaStatus;
//}


// converting functions HOST 

uchar4* converting_UCHAR_UCHAR4(uchar* input ,int size_4) {
    uchar4* output = (uchar4*)malloc(size_4 * sizeof(uchar4));
    memset(output, 100, size_4 * sizeof(uchar4));
    
    for (int i = 0,j=0;j < size_4; i+=4,j++) {
        output[j].x = input[i];
        output[j].y = input[i+1];
        output[j].z = input[i+2];
        output[j].w = input[i+3];
        //cout << "i = " << i << ", j = "<< j << endl;
    }

    return output;
}

uchar* converting_UCHAR4_UCHAR(uchar4* input, int size_1) {
    uchar* output = (uchar*)malloc(size_1 * sizeof(uchar));
    memset(output, 100, size_1 * sizeof(uchar));
    
    for (int i = 0, j = 0; i < size_1; i += 4, j++) {
        output[j] = input[i].x;
        output[j+1] = input[i].y;
        output[j+2] = input[i].z;
        output[j+3] = input[i].w;
        //cout << "i = " << i << ", j = " << j << endl;
    }

    return output;
}

void ucharToMat(uchar* p2,int rows,int cols){
    Mat greyImg = Mat(rows, cols, CV_8U, p2);
    string greyArrWindow = "Grey Array Image";
    namedWindow(greyArrWindow, cv::WINDOW_AUTOSIZE);
    imshow(greyArrWindow, greyImg);

    

    waitKey(0);
    destroyAllWindows();
}


//Display Functions HOST

void DisplayUchar4(uchar4* arr,int size) {
    printf("THIS IS THE UCHAR4 DISPLAY !! \n");
    for (int i = 0; i < size; i++) {
        printf("arr[i].x = %d , arr[i].y = %d , arr[i].z = %d \n", arr[i].x, arr[i].y, arr[i].z);   
    }
}

void DisplayUchar(uchar* arr, int size)
{
    printf("THIS IS THE UCHAR DISPLAY !!\n");
    // small display
    for (int i = 0; i < size; i++) {
        printf("imgData[i] = %d | ", arr[i]);
    }
    cout << endl;
}


// KERNEL FUNCTIONS DEVICE

__global__ void addKernel(uchar4* a, uchar* b)
{
    int blockId = blockIdx.x + blockIdx.y * gridDim.x;
    int threadId = blockId * (blockDim.x * blockDim.y)
        + (threadIdx.y * blockDim.x) + threadIdx.x;

    b[threadId] = .114f * a[threadId].x + .587f * a[threadId].y + .299f * a[threadId].z;
}
