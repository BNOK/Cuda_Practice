#include "hip/hip_runtime.h"
﻿
// OpenCV libs
#include <opencv2/opencv.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/core/core.hpp>
// CUDA libs
#include "hip/hip_runtime.h"
#include ""
// C/C++ libs
#include <stdio.h>
#include <iostream>
#include "utils.h"

#define TILE 16


//namespaces
using namespace std;
using namespace cv;

// FUNCTIONS DEFINITION 
void PreProcess(string path_file, uchar4** h_inputimage, uchar4** h_outputimage, uchar4** d_inputimage, uchar4** d_outputimage, uchar* d_red, uchar* d_green, uchar* d_blue, int* rows,int* cols);
void AllocatingMemy(uchar* arr1, uchar* arr2, uchar* arr3, int size);
// KERNEL DEFINITION
__global__ void display(uchar* arr);
__global__ void Bluring_Kernel();
__global__ void Seperate_Colors(const uchar4* const inputImageRGBA,
    int numRows,
    int numCols,
    unsigned char* const redChannel,
    unsigned char* const greenChannel,
    unsigned char* const blueChannel);


int main()
{
    string path_file = "C:/Users/Mega-Pc/Desktop/git-project/Cuda_Practice/Cuda_Blurring_Effect/Test_Image.png";
    uchar4* h_inputImageRGBA, * d_inputImageRGBA;
    uchar4* h_outputImageRGBA, * d_outputImageRGBA;
    int rows, cols;
    uchar* d_red = 0, * d_green = 0, * d_blue = 0;
    
    PreProcess(path_file, &h_inputImageRGBA, &h_outputImageRGBA, &d_inputImageRGBA, &d_outputImageRGBA, d_red, d_green, d_blue, &rows, &cols);
    printf("hello 1");
    display << <1, 1 >> > (d_red);

    uchar* d_redBlurred, * d_greenBlurred, * d_blueBlurred;

    
   

    //printf("rows = %d , cols = %d", rows, cols);
    //// kernel configuration
    //dim3 BlockDim(TILE, TILE, 1);
    //dim3 GridDim(cols/TILE, rows/TILE, 1);

    //Seperate_Colors << <GridDim, BlockDim >> > (h_inputImageRGBA, rows, cols, d_red, d_green, d_blue);

    waitKey(0);

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
//{
//    int *dev_a = 0;
//    int *dev_b = 0;
//    int *dev_c = 0;
//    hipError_t cudaStatus;
//
//    // Choose which GPU to run on, change this on a multi-GPU system.
//    cudaStatus = hipSetDevice(0);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
//        goto Error;
//    }
//
//    // Allocate GPU buffers for three vectors (two input, one output)    .
//    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    // Copy input vectors from host memory to GPU buffers.
//    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//    // Launch a kernel on the GPU with one thread for each element.
//    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);
//
//    // Check for any errors launching the kernel
//    cudaStatus = hipGetLastError();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
//        goto Error;
//    }
//    
//    // hipDeviceSynchronize waits for the kernel to finish, and returns
//    // any errors encountered during the launch.
//    cudaStatus = hipDeviceSynchronize();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//        goto Error;
//    }
//
//    // Copy output vector from GPU buffer to host memory.
//    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//Error:
//    hipFree(dev_c);
//    hipFree(dev_a);
//    hipFree(dev_b);
//    
//    return cudaStatus;
//}

// utils functions 
void PreProcess(string path_file, uchar4** h_inputimage,uchar4** h_outputimage, uchar4** d_inputimage, uchar4** d_outputimage, uchar* d_red, uchar* d_green, uchar* d_blue,int* rows, int* cols) {
    
    // reading Image
    Mat img = imread(path_file, IMREAD_COLOR);
    Mat rgbaImg;
    cvtColor(img, rgbaImg, COLOR_BGR2RGBA);
    *rows = rgbaImg.rows;
    *cols = rgbaImg.cols;
    int numPixels = rgbaImg.rows * rgbaImg.cols;

    *h_inputimage = (uchar4*)rgbaImg.ptr<unsigned char>(0);
    *h_outputimage = (uchar4*)rgbaImg.ptr<unsigned char>(0);
    

    checkCudaErrors(hipMalloc(d_inputimage, sizeof(uchar4) * numPixels));
    checkCudaErrors(hipMalloc(d_outputimage, sizeof(uchar4) * numPixels));
    checkCudaErrors( hipMemset(*d_outputimage, 0, numPixels * sizeof(uchar4)));

    AllocatingMemy(d_red, d_green, d_blue, numPixels);
}

void AllocatingMemy(uchar* arr1, uchar* arr2, uchar* arr3,int size) {
    int BYTE_SIZE = size * sizeof(uchar);

    checkCudaErrors(hipMalloc(&arr1, BYTE_SIZE));
    checkCudaErrors(hipMalloc(&arr2, BYTE_SIZE));
    checkCudaErrors(hipMalloc(&arr3, BYTE_SIZE));

    checkCudaErrors(hipMemset(arr1, 1, BYTE_SIZE));
    checkCudaErrors(hipMemset(arr2, 2, BYTE_SIZE));
    checkCudaErrors(hipMemset(arr3, 3, BYTE_SIZE));
}
// KERNEL IMPLEMENTATION
__global__ void display(uchar* arr) {
    cout << "hello world !" << endl;
    int threadId = threadIdx.x;
    int element = arr[threadId];
    cout<<"arr[i] = % d" << element<<endl;
}

__global__ void Bluring_Kernel()
{

}

__global__ void Seperate_Colors(const uchar4* const inputImageRGBA,
    int numRows,
    int numCols,
    unsigned char* const redChannel,
    unsigned char* const greenChannel,
    unsigned char* const blueChannel) {

    //thread index x and y 
    int Xid = blockIdx.x * blockDim.x + threadIdx.x;
    int Yid = blockIdx.y * blockDim.y + threadIdx.y;
    // thread index 
    int blockId = blockIdx.x + blockIdx.y * gridDim.x;
    int threadId = blockId * (blockDim.x * blockDim.y)
        + (threadIdx.y * blockDim.x) + threadIdx.x;

    if (Xid > numRows || Yid > numCols)
        return;

    redChannel[threadId] = inputImageRGBA[threadId].x;
    greenChannel[threadId] = inputImageRGBA[threadId].y;
    blueChannel[threadId] = inputImageRGBA[threadId].z;

    
}

void DisplayArray(unsigned char* arr, int size) {
    for (int i = 0; i < size; i++) {
        printf("arr[i] = %d | ", arr[i]);
    }
}
